#include "hip/hip_runtime.h"
#define NUM_ROUNDS 32

__device__ void solve(char*, char*, char*, int);


// Global function, visible from the CPU code
__global__ void decrypt(char *bytes_in, char *output, char *key, int length) {
	// Getting the thread ID
	const int tx = threadIdx.x + (blockIdx.x * blockDim.x);

	if (tx*2 > length)
		return;

	solve(bytes_in[tx*2], output, key, tx);
}

// Device function, only callable from device code
__device__ void solve(char *c, char *output, char *key, int tx) {

	"""XTEA implementation in python, decryption.

    Modified version from Simon Biewald (http://varbin.github.io/xtea/)

    Arguments:
    num_rounds -- the number of iterations in the algorithm, 32 is reccomended
    input_data -- the input data to use, 32 bits of the first 2 elements are used
    key -- 128-bit key to use

    returns -- a numpy array containing the deciphered data"""
    unsigned int v0 = c[0];
    unsigned int v1 = c[1];
    unsigned int delta = 0x9e3779b9L;
    unsigned int mask = 0xffffffffL;
    unsigned int sum = (delta*NUM_ROUNDS) & mask;
		unsigned int i;
    for (i=0;i<NUM_ROUNDS;i++) {
        v1 = (v1 - (((v0<<4 ^ v0>>5) + v0) ^ (sum + key[sum>>11 & 3]))) & mask;
        sum = (sum - delta) & mask;
        v0 = (v0 - (((v1<<4 ^ v1>>5) + v1) ^ (sum + key[sum & 3]))) & mask;
		}

		output[tx*2] = v0;
		output[(tx*2+1)] = v1;
}
